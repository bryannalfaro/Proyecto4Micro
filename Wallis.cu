//Universidad del Valle de Guatemala
//Proyecto 4 - Calculo de Pi con Wallis
//Programacion de Microprocesadores
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void
serieWallis(float *convergencia, int *vectorN, int limite)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    float operacion;
    float operacion2;
 
    if((i-1)<=limite){
        operacion = (2.0f*(vectorN[i-1]))/((2.0f*(vectorN[i-1]))-1.0f);
        operacion2 = (2.0f*(i))/((2.0f*(i))+1.0f);
        convergencia[i]=operacion*operacion2;
    }
}

int main(void){

    int valorN =1000;
    float result=1.0f;
    size_t sizef = 1000*valorN* sizeof(float);
    size_t sizei = 1000*valorN* sizeof(int);

    //Reserva en el host
    int *host_vectorN= (int *)malloc(sizei);
    float *host_vectorValor= (float *)malloc(sizef);

    //Llenado con los valores de N
    for(int i = 1; i <= valorN; i++)
     {
         host_vectorN[i-1]=i;
     }

     for(int i = 1; i <= valorN; i++)
     {
         //printf("Valor: %d",host_vectorN[3]);
     }

    int *d_vectorN = NULL;
    hipMalloc((void **)&d_vectorN,sizei);
    float *d_vectorValor = NULL;
    hipMalloc((void **)&d_vectorValor, sizef);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipMemcpy(d_vectorN, host_vectorN,sizei, hipMemcpyHostToDevice);
    hipMemcpy(d_vectorValor, host_vectorValor, sizef, hipMemcpyHostToDevice);


    int threadsPerBlock = 1000;
    //int blocksPerGrid =(threadsPerBlock+1) /;
    int blocksPerGrid =(valorN + threadsPerBlock - 1) / threadsPerBlock;
    
    printf("%d\n",blocksPerGrid);
    
    hipEventRecord(start);
    serieWallis<<<blocksPerGrid+1,threadsPerBlock>>>(d_vectorValor, d_vectorN, valorN);
    hipEventRecord(stop);

    hipMemcpy(host_vectorValor, d_vectorValor, sizef, hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    

    
    printf("Milisegundos: %.5f\n",milliseconds);
    printf("Segundos: %.5f\n",milliseconds/1000);

    for(int j=1;j<=valorN;j++){
        //printf("\nEl valor  es:  %.7f\n",host_vectorValor[j]);  
        result*=(host_vectorValor[j]);
       // printf("\nEl result  es:  %.7f\n",result);
         
     }

     printf("\nEl valor pi es:  %.16f\n",result*2);

    free(host_vectorN);
    free(host_vectorValor);
    
    hipFree(d_vectorN);
    hipFree(d_vectorValor);

}