#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
__global__ void nila(double *vector_2, double *vector_suma, int n)
{
    #include <math.h>
    // identificador de hilo
    int myID = threadIdx.x;
    int myid2 = (threadIdx.x +1)*2.f;
    if((threadIdx.x +1)%2 == 0)
    {
        vector_2[myID] = (4.f/(myid2*(myid2+1.f)*(myid2+2.f)))*-1.f;
    }else{
        vector_2[myID] = 4.f/(myid2*(myid2+1.f)*(myid2+2.f));  
    }
    
    // escritura de resultados
    vector_suma[myID] = vector_2[myID];
}

int main(int argc, char** argv)
{
    // declaraciones
    double *hst_vector2, *hst_resultado;
    double *dev_vector2, *dev_resultado;
    int n = 1000;
    
    // reserva en el host
    hst_vector2 = (double*)malloc(n * sizeof(double));
    hst_resultado = (double*)malloc(n * sizeof(double));

    // reserva en el device
    hipMalloc((void**)&dev_vector2, n * sizeof(double));
    hipMalloc((void**)&dev_resultado, n * sizeof(double));

    // inicializacion de vectores
    for (int i = 0; i < n; i++)
    {
        hst_vector2[i] = 0;
    }
    
    // LANZAMIENTO DEL KERNEL
    nila << < 1, n >> >(dev_vector2, dev_resultado, n);

    // recogida de datos desde el device
    hipMemcpy(hst_vector2, dev_vector2, n * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(hst_resultado, dev_resultado, n * sizeof(double), hipMemcpyDeviceToHost);

    
    // impresion de resultados
    double suma = 0;
    for (int i = 0; i < n; i++)
    {
        suma += hst_resultado[i]*1.f;
    }
    printf("Pi con la serie de Nilakantha: %2f ", suma+3.f);
    printf("\n");
    // salida

    return 0;
}