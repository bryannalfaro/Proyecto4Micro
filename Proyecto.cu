//Universidad del Valle de Guatemala
//Proyecto 4 - Calculo de Pi con Wallis y Nilakantha
//Programacion de Microprocesadores
//Integrantes:
//Bryann Alfaro 19372
//Diego Arredondo 19422
//Donaldo Garcia 19683
//Raul Jimenez 19017
//Diego Alvarez 19498

#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void serieWallis(float *convergencia, int *vectorN, int limite)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    float operacion;
    float operacion2;
 
    if((i-1)<=limite){
        operacion = (2.0f*(vectorN[i-1]))/((2.0f*(vectorN[i-1]))-1.0f);
        operacion2 = (2.0f*(i))/((2.0f*(i))+1.0f);
        convergencia[i]=operacion*operacion2;
    }
}

__global__ void nila(double *vector_2, double *vector_suma, int n)
{
    #include <math.h>
    // identificador de hilo
    int myID = threadIdx.x;
    int myid2 = (threadIdx.x +1)*2.f;
    if((threadIdx.x +1)%2 == 0)
    {
        vector_2[myID] = (4.f/(myid2*(myid2+1.f)*(myid2+2.f)))*-1.f;
    }else{
        vector_2[myID] = 4.f/(myid2*(myid2+1.f)*(myid2+2.f));  
    }
    
    // escritura de resultados
    vector_suma[myID] = vector_2[myID];
}

int main(void){

    //Inicializacion de Streams
    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    int valorN =1000;

    //Stream 1
    float result=1.0f;
    size_t sizef = 10*valorN* sizeof(float);
    size_t sizei = 10*valorN* sizeof(int);

    //Stream 2
    double *hst_vector2, *hst_resultado;
    double *dev_vector2, *dev_resultado;

    //Reserva en el host Stream 1
    int *host_vectorN= (int *)malloc(sizei);
    float *host_vectorValor= (float *)malloc(sizef);

    //Reserva en el device Stream 1
    int *d_vectorN = NULL;
    hipMalloc((void **)&d_vectorN,sizei);
    float *d_vectorValor = NULL;
    hipMalloc((void **)&d_vectorValor, sizef);

    //Reserva en el host Stream 2
    hst_vector2 = (double*)malloc(valorN * sizeof(double));
    hst_resultado = (double*)malloc(valorN * sizeof(double));

    //Reserva en el device Stream 2
    hipMalloc((void**)&dev_vector2, valorN * sizeof(double));
    hipMalloc((void**)&dev_resultado, valorN * sizeof(double));

    //Llenado con los valores de N
    for(int i = 1; i <= valorN; i++)
     {
          host_vectorN[i-1]=i;
          hst_vector2[i] = 0;
     }


    //Creacion de Evento para tomar el tiempo
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    //Bloques e hilos para la ejecucion de los kernels
    int threadsPerBlock = 1000;
    int blocksPerGrid = (valorN + threadsPerBlock - 1) / threadsPerBlock;


    //Stream 1
    hipMemcpyAsync(d_vectorN, host_vectorN,sizei, hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(d_vectorValor, host_vectorValor, sizef, hipMemcpyHostToDevice, stream1);

    hipEventRecord(start);
    serieWallis<<<blocksPerGrid+1,threadsPerBlock, 0, stream1>>>(d_vectorValor, d_vectorN, valorN);
    hipEventRecord(stop);

    hipMemcpyAsync(host_vectorValor, d_vectorValor, sizef, hipMemcpyDeviceToHost, stream1);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);


    //Stream 2
    hipMemcpyAsync(dev_vector2, hst_vector2, valorN * sizeof(double), hipMemcpyHostToDevice, stream2);
    hipMemcpyAsync(hst_resultado, dev_resultado, valorN * sizeof(double), hipMemcpyHostToDevice, stream2);
    hipEventRecord(start);
    nila <<< blocksPerGrid+1,threadsPerBlock, 0, stream2 >>>(dev_vector2, dev_resultado, valorN);
    hipEventRecord(stop);
    hipMemcpyAsync(hst_resultado, dev_resultado, valorN * sizeof(double), hipMemcpyDeviceToHost, stream2);

    hipEventSynchronize(stop);
    float milliseconds2 = 0;
    hipEventElapsedTime(&milliseconds2, start, stop);

    //Sincronizar Streams
    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);


    //Impresion de Datos del Stream 1

    printf("--------Stream 1--------\n");

    printf("Milisegundos: %.5f\n",milliseconds);
    printf("Segundos: %.5f\n",milliseconds/1000);

    for(int j=1;j<=valorN;j++){
        
        result*=(host_vectorValor[j]);

     }

    printf("\nPi con la serie de Wallis:  %.16f\n\n",result*2);


    //Impresion de Datos del Stream 2

    printf("--------Stream 2--------\n");
     
    printf("Milisegundos: %.5f\n",milliseconds2);
    printf("Segundos: %.5f\n",milliseconds2/1000);

    double suma = 0;
    for (int i = 0; i < valorN; i++)
    {
        suma += hst_resultado[i]*1.f;
    }
    printf("\nPi con la serie de Nilakantha: %.16f ", suma+3.f);
    printf("\n");

    //Salida

    //Destruccion de Streams
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);

    //Liberacion de host
    free(host_vectorN);
    free(host_vectorValor);
    
    //Liberacion de Device
    hipFree(d_vectorN);
    hipFree(d_vectorValor);
}